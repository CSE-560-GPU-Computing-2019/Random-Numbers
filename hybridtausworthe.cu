
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <sstream>
using namespace std;

typedef unsigned long long llu;

int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

#define MT_RNG_COUNT 2

// int    PATH_N = 1000000;
// int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
// int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

// __global__ void RandomCT(llu *device_array, int npr) {
//     const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
//     llu seed = 10 * tid + 5;

//     llu b;
//     for (int i = 0; i < npr; ++i) {
//         b = (((seed << 5) ^ seed) >> 39);
//         seed = (((seed & 18446744073709551614ULL) << 24) ^ b);

//         device_array[tid + i * MT_RNG_COUNT] = seed;
//     }

// }

typedef struct {
    llu s1, s2, s3, s4;
} tauswortheState;


__device__ void lcg(llu *s, llu p, llu a, llu b) {
    *s = a * p + b;
}

__global__ void RandomHT(llu *device_array, int npr) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    llu s1, s2, s3, s4;
    llu seed = 10 * tid + 129;
    
    lcg(&s1, seed, 1664525, 1013904223ULL);
    lcg(&s2, s1, 1664525, 1013904223ULL);
    lcg(&s3, s2, 1664525, 1013904223ULL);
    lcg(&s4, s3, 1664525, 1013904223ULL);
    
    llu b;
    for (int i = 0; i < 4; ++i) {
        b = (((s1 << 5) ^ s1) >> 39);
        s1 = (((s1 & 18446744073709551614ULL) << 24) ^ b);

        b = (((s2 << 19) ^ s2) >> 45);
        s2 = (((s2 & 18446744073709551552ULL) << 13) ^ b);

        b = (((s3 << 24) ^ s3) >> 48);
        s3 = (((s3 & 18446744073709551104ULL) << 7) ^ b);

        // s4 = lcg(s4, 1664525, 1013904223ULL);
        lcg(&s4, s4, 1664525, 1013904223ULL);
    }
    
    for (int i = 0; i < npr; ++i) {
        b = (((s1 << 5) ^ s1) >> 39);
        s1 = (((s1 & 18446744073709551614ULL) << 24) ^ b);

        b = (((s2 << 19) ^ s2) >> 45);
        s2 = (((s2 & 18446744073709551552ULL) << 13) ^ b);

        b = (((s3 << 24) ^ s3) >> 48);
        s3 = (((s3 & 18446744073709551104ULL) << 7) ^ b);

        lcg(&s4, s4, 1664525, 1013904223ULL);
        
        b = s1 ^ s2 ^ s3 ^ s4;
        device_array[tid + i * MT_RNG_COUNT] = b;
    }
    
}

int main() {
    for (int xx = 10; xx <= 10; xx *= 10) {
        ostringstream os;
        os << "RANDOMNUMBERS_" << xx << "_HT_GPU.txt";
        string x = os.str();
        // freopen(x.c_str(), "w", stdout);

        // freopen("HybridParallel.txt", "w", stdout);
        // for (int i = 10000; i <= 1000000000; i *= 10) {
        //     int    PATH_N = i;
        //     int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
        //     int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

        //     printf("%d ", RAND_N);
        // }
        // printf("\n");
        // float ktime[6], tTime[6];

        // for (int i = 10000, j = 0; i <= 1000000000, j < 6; i *= 10, j++) {
            int    PATH_N = xx;
            int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
            int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

            // cudaEvent_t start, stop, memstart, memstop;
            // cudaEventCreate(&memstart);
            // cudaEventCreate(&memstop);
            // cudaEventCreate(&start);
            // cudaEventCreate(&stop);

            llu *device_array;
            
            llu *host_array;
            llu *host_copy;

            host_array = (llu *)malloc(RAND_N * sizeof(llu));
            host_copy = (llu *)malloc(RAND_N * sizeof(llu));

            // cudaEventRecord(memstart);
            hipMalloc((void**)&device_array, RAND_N * sizeof(llu));

            // int iters = 100;
            // for (int i = 0; )
            // cudaEventRecord(start);
            RandomHT<<<32, 128>>>(device_array, N_PER_RNG);
            // cudaEventRecord(stop);

            hipMemcpy(host_copy, device_array, RAND_N * sizeof(llu), hipMemcpyDeviceToHost);
            // cudaEventRecord(memstop);

            // cudaEventSynchronize(stop);
            // cudaEventSynchronize(memstop);
            
            for (int i = 0; i < RAND_N; ++i) {
                printf("%llu\n", host_copy[i]);
            }

            // float kernelTime, totalTime;
            // cudaEventElapsedTime(&kernelTime, start, stop);
            // cudaEventElapsedTime(&totalTime, memstart, memstop);
            // kernelTime /= 1000.0f;
            // totalTime /= 1000.0f;
            
            // cout << "Time taken for " << RAND_N << " random numbers: \n";
            // cout << "Kernel Execution time: " << kernelTime << "s\n";
            // cout << "Overall Time: " << totalTime << "s\n";
            // ktime[j] = kernelTime;
            // tTime[j] = totalTime;

            free(host_array); free(host_copy);
            hipFree(device_array);
        // }

        // for (int i = 0; i < 6; ++i) 
        //     printf("%f ", ktime[i]);
        // printf("\n");
        // for (int i = 0; i < 6; ++i) 
        //     printf("%f ", tTime[i]);
        // printf("\n");
        
        // cudaEvent_t start, stop, memstart, memstop;
        // cudaEventCreate(&memstart);
        // cudaEventCreate(&memstop);
        // cudaEventCreate(&start);
        // cudaEventCreate(&stop);

        // llu *device_array;
        
        // llu *host_array;
        // llu *host_copy;

        // host_array = (llu *)malloc(RAND_N * sizeof(llu));
        // host_copy = (llu *)malloc(RAND_N * sizeof(llu));

        // cudaEventRecord(memstart);
        // cudaMalloc((void**)&device_array, RAND_N * sizeof(llu));

        // // int iters = 100;
        // // for (int i = 0; )
        // cudaEventRecord(start);
        // RandomHT<<<32, 128>>>(device_array, N_PER_RNG);
        // cudaEventRecord(stop);

        // cudaMemcpy(host_copy, device_array, RAND_N * sizeof(llu), cudaMemcpyDeviceToHost);
        // cudaEventRecord(memstop);

        // cudaEventSynchronize(stop);
        // cudaEventSynchronize(memstop);
        
        // // for (int i = 0; i < 10; ++i) {
        // //     printf("%llu\n", host_copy[i]);
        // // }

        // float kernelTime, totalTime;
        // cudaEventElapsedTime(&kernelTime, start, stop);
        // cudaEventElapsedTime(&totalTime, memstart, memstop);
        // kernelTime /= 1000.0f;
        // totalTime /= 1000.0f;
        
        // cout << "Time taken for " << RAND_N << " random numbers: \n";
        // cout << "Kernel Execution time: " << kernelTime << "s\n";
        // cout << "Overall Time: " << totalTime << "s\n";
        fclose(stdout);
    }

    return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

typedef unsigned long long llu;

int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

#define MT_RNG_COUNT 4096

const int    PATH_N = 24000000;
const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

// __global__ void RandomCT(llu *device_array, int npr) {
//     const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
//     llu seed = 10 * tid + 5;

//     llu b;
//     for (int i = 0; i < npr; ++i) {
//         b = (((seed << 5) ^ seed) >> 39);
//         seed = (((seed & 18446744073709551614ULL) << 24) ^ b);

//         device_array[tid + i * MT_RNG_COUNT] = seed;
//     }

// }

typedef struct {
    llu s1, s2, s3, s4;
} tauswortheState;


__device__ void lcg(llu *s, llu p, llu a, llu b) {
    *s = a * p + b;
}

__global__ void RandomHT(llu *device_array, int npr) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    llu s1, s2, s3, s4;
    llu seed = 10 * tid + 129;
    
    lcg(&s1, seed, 1664525, 1013904223ULL);
    lcg(&s2, s1, 1664525, 1013904223ULL);
    lcg(&s3, s2, 1664525, 1013904223ULL);
    lcg(&s4, s3, 1664525, 1013904223ULL);
    
    llu b;
    for (int i = 0; i < 4; ++i) {
        b = (((s1 << 5) ^ s1) >> 39);
        s1 = (((s1 & 18446744073709551614ULL) << 24) ^ b);

        b = (((s2 << 19) ^ s2) >> 45);
        s2 = (((s2 & 18446744073709551552ULL) << 13) ^ b);

        b = (((s3 << 24) ^ s3) >> 48);
        s3 = (((s3 & 18446744073709551104ULL) << 7) ^ b);

        // s4 = lcg(s4, 1664525, 1013904223ULL);
        lcg(&s4, s4, 1664525, 1013904223ULL);
    }
    
    for (int i = 0; i < npr; ++i) {
        b = (((s1 << 5) ^ s1) >> 39);
        s1 = (((s1 & 18446744073709551614ULL) << 24) ^ b);

        b = (((s2 << 19) ^ s2) >> 45);
        s2 = (((s2 & 18446744073709551552ULL) << 13) ^ b);

        b = (((s3 << 24) ^ s3) >> 48);
        s3 = (((s3 & 18446744073709551104ULL) << 7) ^ b);

        lcg(&s4, s4, 1664525, 1013904223ULL);
        
        b = s1 ^ s2 ^ s3 ^ s4;
        device_array[tid + i * MT_RNG_COUNT] = b;
    }
    
}

int main() {
    llu *device_array;
    
    llu *host_array;
    llu *host_copy;

    host_array = (llu *)malloc(RAND_N * sizeof(llu));
    host_copy = (llu *)malloc(RAND_N * sizeof(llu));
    hipMalloc((void**)&device_array, RAND_N * sizeof(llu));

    // int iters = 100;
    // for (int i = 0; )
    RandomHT<<<32, 128>>>(device_array, N_PER_RNG);

    hipMemcpy(host_copy, device_array, RAND_N * sizeof(llu), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < 10; ++i) {
        printf("%llu\n", host_copy[i]);
    }

    return 0;
}
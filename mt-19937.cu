#include "hip/hip_runtime.h"
#include "mt-19937.h"

static mt_struct MT[MT_RNG_COUNT];
static uint32_t state[MT_NN];

const int    PATH_N = 24000000;
const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;
const unsigned int SEED = 777;


int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

int iDivDown(int a, int b)
{
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b)
{
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b)
{
    return a - a % b;
}

/////////////////////////
void initMTRef(const char *fname)
{

    FILE *fd = fopen(fname, "rb");
    if(!fd){
        shrLog("initMTRef(): failed to open %s\n", fname);
        shrLog("FAILED\n");
        exit(0);
    }

    for (int i = 0; i < MT_RNG_COUNT; i++){
        //Inline structure size for compatibility,
        //since pointer types are 8-byte on 64-bit systems (unused *state variable)
        if( !fread(MT + i, 16 /* sizeof(mt_struct) */ * sizeof(int), 1, fd) ){
            shrLog("initMTRef(): failed to load %s\n", fname);
            shrLog("FAILED\n");
            exit(0);
        }
    }

    fclose(fd);
}


void RandomRef(float *h_Random, int NPerRng, unsigned int seed)
{
    int iRng, iOut;

    for(iRng = 0; iRng < MT_RNG_COUNT; iRng++){
        MT[iRng].state = state;
        sgenrand_mt(seed, &MT[iRng]);

        for(iOut = 0; iOut < NPerRng; iOut++)
           h_Random[iRng * NPerRng + iOut] = ((float)genrand_mt(&MT[iRng]) + 1.0f) / 4294967296.0f;
    }
}


static void BoxMuller(float& u1, float& u2)
{
    float   r = sqrtf(-2.0f * logf(u1));
    float phi = 2 * PI * u2;
    u1 = r * cosf(phi);
    u2 = r * sinf(phi);
}


void BoxMullerRef(float *h_Random, int NPerRng)
{
    int i;

    for(i = 0; i < MT_RNG_COUNT * NPerRng; i += 2)
        BoxMuller(h_Random[i + 0], h_Random[i + 1]);
}


//////////////////////////
void loadMTGPU(const char *fname)
{
    FILE *fd = fopen(fname, "rb");
    if(!fd){
        shrLog("initMTGPU(): failed to open %s\n", fname);
        shrLog("FAILED\n");
        exit(0);
    }
    if( !fread(h_MT, sizeof(h_MT), 1, fd) ){
        shrLog("initMTGPU(): failed to load %s\n", fname);
        shrLog("FAILED\n");
        exit(0);
    }
    fclose(fd);
}


void seedMTGPU(unsigned int seed){
    int i;
    //Need to be thread-safe
    mt_struct_stripped *MT = (mt_struct_stripped *)malloc(MT_RNG_COUNT * sizeof(mt_struct_stripped));

    for(i = 0; i < MT_RNG_COUNT; i++){
        MT[i]      = h_MT[i];
        MT[i].seed = seed;
    }
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(ds_MT), MT, sizeof(h_MT)) );

    free(MT);
}


__global__ void gpuRand(float *d_Random, int nPerRng)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    int iState, iState1, iStateM, iOut;
    unsigned int mti, mti1, mtiM, x;
    unsigned int mt[MT_NN], matrix_a, mask_b, mask_c;

    //Load bit-vector Mersenne Twister parameters
    matrix_a = ds_MT[tid].matrix_a;
    mask_b = ds_MT[tid].mask_b;
    mask_c = ds_MT[tid].mask_c;

    //Initialize current state
    mt[0] = ds_MT[tid].seed;
    for (iState = 1; iState < MT_NN; iState++)
        mt[iState] = (1812433253U * (mt[iState - 1] ^ (mt[iState - 1] >> 30)) + iState) & MT_WMASK;

    iState = 0;
    mti1 = mt[0];
    for (iOut = 0; iOut < nPerRng; iOut++)
    {
        iState1 = iState + 1;
        iStateM = iState + MT_MM;
        if(iState1 >= MT_NN) iState1 -= MT_NN;
        if(iStateM >= MT_NN) iStateM -= MT_NN;
        mti  = mti1;
        mti1 = mt[iState1];
        mtiM = mt[iStateM];

        // MT recurrence
        x    = (mti & MT_UMASK) | (mti1 & MT_LMASK);
        x    =  mtiM ^ (x >> 1) ^ ((x & 1) ? matrix_a : 0);

        mt[iState] = x;
        iState = iState1;

        //Tempering transformation
        x ^= (x >> MT_SHIFT0);
        x ^= (x << MT_SHIFTB) & mask_b;
        x ^= (x << MT_SHIFTC) & mask_c;
        x ^= (x >> MT_SHIFT1);

        //Convert to (0, 1] float and write to global memory
        d_Random[tid + iOut * MT_RNG_COUNT] = ((float)x + 1.0f) / 4294967296.0f;
    }
}


__device__ inline void BoxMuller(float& u1, float& u2)
{
    float   r = sqrtf(-2.0f * logf(u1));
    float phi = 2 * PI * u2;
    u1 = r * __cosf(phi);
    u2 = r * __sinf(phi);
}


__global__ void BoxMullerGPU(float *d_Random, int nPerRng)
{
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;

    for (int iOut = 0; iOut < nPerRng; iOut += 2)
        BoxMuller(
                d_Random[tid + (iOut + 0) * MT_RNG_COUNT],
                d_Random[tid + (iOut + 1) * MT_RNG_COUNT]
                );
}


int main()
{
    FILE *log_file;
    log_file = fopen("mt-19937.txt", "w"); 

    float *d_rand_out, *h_randCPU_out, *h_randGPU_out;

    //Allocating memory
    h_randCPU_out  = (float *)malloc(RAND_N * sizeof(float));
    h_randGPU_out  = (float *)malloc(RAND_N * sizeof(float));
    hipMalloc((void **)&d_rand_out, RAND_N * sizeof(float))


    fprintf(log_file, "Loading CPU and GPU twisters configurations...\n");
    initMTRef('data/MersenneTwister.raw');
    loadMTGPU('data/MersenneTwister.dat');
    seedMTGPU(SEED);

    cutCreateTimer(&hTimer)

    int numIterations = 100;
	for (int i = -1; i < numIterations; i++)
	{
		if (i == 0)
		{
			hipDeviceSynchronize();
			cutResetTimer(hTimer);
			cutStartTimer(hTimer);
		}
	RandomGPU<<<32, 128>>>(d_Rand, N_PER_RNG);
    #ifdef DO_BOXMULLER
    BoxMullerGPU<<<32, 128>>>(d_Rand, N_PER_RNG);
    #endif
    }

    hipDeviceSynchronize()

    fprintf(log_file, "MersenneTwister, Throughput = %.4f GNumbers/s, Time = %.5f s, Size = %u Numbers, NumDevsUsed = %u, Workgroup = %u\n", 1.0e-9 * RAND_N / gpuTime, gpuTime, RAND_N, 1, 128);
    hipMemcpy(h_randGPU_out, d_rand_out, RAND_N * sizeof(float), hipMemcpyDeviceToHost)

    //time this
    RandomRef(h_RandCPU, N_PER_RNG, SEED);
    #ifdef DO_BOXMULLER
    BoxMullerRef(h_RandCPU, N_PER_RNG);
    #endif

    return 0;
}
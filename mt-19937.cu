#include "hip/hip_runtime.h"
#include <shrUtils.h>

void loadMTGPU(const char *fname)
{
    FILE *fd = fopen(fname, "rb");
    if(!fd){
        shrLog("initMTGPU(): failed to open %s\n", fname);
        shrLog("FAILED\n");
        exit(0);
    }
    if( !fread(h_MT, sizeof(h_MT), 1, fd) ){
        shrLog("initMTGPU(): failed to load %s\n", fname);
        shrLog("FAILED\n");
        exit(0);
    }
    fclose(fd);
}

__global__ void gpuRand()
{

}


int main()
{
    shrSetLogFileName ("mt-19337.txt");

    float *d_rand_out, *h_rand_out;

    return 0;
}
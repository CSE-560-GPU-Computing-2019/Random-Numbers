#include "hip/hip_runtime.h"
#include "mt-19937.h"
#include <stdio.h>
#include <stdint.h>
<<<<<<< HEAD
// #include <shrUtils.h>
=======
>>>>>>> bca7277f2af986aa2b908854358b226236f595a9

#define DCMT_SEED 4172
#define MT_RNG_PERIOD 607
#define MT_RNG_COUNT 4096
#define MT_MM 9
#define MT_NN 19
#define MT_WMASK 0xFFFFFFFFU
#define MT_UMASK 0xFFFFFFFEU
#define MT_LMASK 0x1U
#define MT_SHIFT0 12
#define MT_SHIFTB 7
#define MT_SHIFTC 15
#define MT_SHIFT1 18
#define PI 3.14159265358979f

#define SHIFT1 18


typedef struct {
    uint32_t aaa;
    int mm,nn,rr,ww;
    uint32_t wmask,umask,lmask;
    int shift0, shift1, shiftB, shiftC;
    uint32_t maskB, maskC;
    int i;
    uint32_t *state;
}mt_struct;


typedef struct{
    unsigned int matrix_a;
    unsigned int mask_b;
    unsigned int mask_c;
    unsigned int seed;
} mt_struct_stripped;

static mt_struct MT[MT_RNG_COUNT];
static uint32_t state[MT_NN];



void sgenrand_mt(uint32_t seed, mt_struct *mts){
    int i;

    mts->state[0] = seed & mts->wmask;

    for(i = 1; i < mts->nn; i++){
        mts->state[i] = (UINT32_C(1812433253) * (mts->state[i - 1] ^ (mts->state[i - 1] >> 30)) + i) & mts->wmask;
        /* See Knuth TAOCP Vol2. 3rd Ed. P.106 for multiplier. */
        /* In the previous versions, MSBs of the seed affect   */
        /* only MSBs of the array mt[].                        */
    }
    mts->i = mts->nn;
}


uint32_t genrand_mt(mt_struct *mts){
    uint32_t *st, uuu, lll, aa, x;
    int k,n,m,lim;

    if(mts->i >= mts->nn ){
        n = mts->nn; m = mts->mm;
        aa = mts->aaa;
        st = mts->state;
        uuu = mts->umask; lll = mts->lmask;

        lim = n - m;
        for(k = 0; k < lim; k++){
            x = (st[k]&uuu)|(st[k+1]&lll);
            st[k] = st[k + m] ^ (x >> 1) ^ (x&1U ? aa : 0U);
        }

        lim = n - 1;
        for(; k < lim; k++){
            x = (st[k] & uuu)|(st[k + 1] & lll);
            st[k] = st[k + m - n] ^ (x >> 1) ^ (x & 1U ? aa : 0U);
        }

        x = (st[n - 1] & uuu)|(st[0] & lll);
        st[n - 1] = st[m - 1] ^ (x >> 1) ^ (x&1U ? aa : 0U);
        mts->i=0;
    }

    x = mts->state[mts->i];
    mts->i += 1;
    x ^= x >> mts->shift0;
    x ^= (x << mts->shiftB) & mts->maskB;
    x ^= (x << mts->shiftC) & mts->maskC;
    x ^= x >> mts->shift1;

    return x;
}



int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

int iDivDown(int a, int b)
{
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b)
{
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b)
{
    return a - a % b;
}


const int    PATH_N = 24000000;
const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;
const unsigned int SEED = 777;

__device__ static mt_struct_stripped ds_MT[MT_RNG_COUNT];
static mt_struct_stripped h_MT[MT_RNG_COUNT];

/////////////////////////
void initMTRef(const char *fname)
{

    FILE *fd = fopen(fname, "rb");
    if(!fd)
    {
        exit(0);
    }

    for (int i = 0; i < MT_RNG_COUNT; i++){
        //Inline structure size for compatibility,
        //since pointer types are 8-byte on 64-bit systems (unused *state variable)
        if( !fread(MT + i, 16 /* sizeof(mt_struct) */ * sizeof(int), 1, fd) )
        {
            exit(0);
        }
    }

    fclose(fd);
}


void RandomRef(float *h_Random, int NPerRng, unsigned int seed)
{
    int iRng, iOut;

    for(iRng = 0; iRng < MT_RNG_COUNT; iRng++){
        MT[iRng].state = state;
        sgenrand_mt(seed, &MT[iRng]);

        for(iOut = 0; iOut < NPerRng; iOut++)
           h_Random[iRng * NPerRng + iOut] = ((float)genrand_mt(&MT[iRng]) + 1.0f) / 4294967296.0f;
    }
}


//////////////////////////
void loadMTGPU(const char *fname)
{
    FILE *fd = fopen(fname, "rb");
    if(!fd)
    {
        exit(0);
    }
    if( !fread(h_MT, sizeof(h_MT), 1, fd) )
    {
        exit(0);
    }
    fclose(fd);
}


void seedMTGPU(unsigned int seed){
    int i;
    //Need to be thread-safe
    mt_struct_stripped *MT = (mt_struct_stripped *)malloc(MT_RNG_COUNT * sizeof(mt_struct_stripped));

    for(i = 0; i < MT_RNG_COUNT; i++){
        MT[i]      = h_MT[i];
        MT[i].seed = seed;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(ds_MT), MT, sizeof(h_MT));

    free(MT);
}


__global__ void gpuRand(float *d_Random, int nPerRng)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    int iState, iState1, iStateM, iOut;
    unsigned int mti, mti1, mtiM, x;
    unsigned int mt[MT_NN], matrix_a, mask_b, mask_c;

    //Load bit-vector Mersenne Twister parameters
    matrix_a = ds_MT[tid].matrix_a;
    mask_b = ds_MT[tid].mask_b;
    mask_c = ds_MT[tid].mask_c;

    //Initialize current state
    mt[0] = ds_MT[tid].seed;
    for (iState = 1; iState < MT_NN; iState++)
        mt[iState] = (1812433253U * (mt[iState - 1] ^ (mt[iState - 1] >> 30)) + iState) & MT_WMASK;

    iState = 0;
    mti1 = mt[0];
    for (iOut = 0; iOut < nPerRng; iOut++)
    {
        iState1 = iState + 1;
        iStateM = iState + MT_MM;
        if(iState1 >= MT_NN) iState1 -= MT_NN;
        if(iStateM >= MT_NN) iStateM -= MT_NN;
        mti  = mti1;
        mti1 = mt[iState1];
        mtiM = mt[iStateM];

        // MT recurrence
        x    = (mti & MT_UMASK) | (mti1 & MT_LMASK);
        x    =  mtiM ^ (x >> 1) ^ ((x & 1) ? matrix_a : 0);

        mt[iState] = x;
        iState = iState1;

        //Tempering transformation
        x ^= (x >> MT_SHIFT0);
        x ^= (x << MT_SHIFTB) & mask_b;
        x ^= (x << MT_SHIFTC) & mask_c;
        x ^= (x >> MT_SHIFT1);

        //Convert to (0, 1] float and write to global memory
        d_Random[tid + iOut * MT_RNG_COUNT] = ((float)x + 1.0f) / 4294967296.0f;
    }
}


int main()
{
    FILE *log_file;
    log_file = fopen("mt-19937.txt", "w"); 

    float *d_rand_out, *h_randCPU_out, *h_randGPU_out;

    //Allocating memory
    h_randCPU_out  = (float *)malloc(RAND_N * sizeof(float));
    h_randGPU_out  = (float *)malloc(RAND_N * sizeof(float));
    hipMalloc((void **)&d_rand_out, RAND_N * sizeof(float));

    initMTRef("data/MersenneTwister.raw");
    loadMTGPU("data/MersenneTwister.dat");
    seedMTGPU(SEED);

    float hTimer;
    hipEvent_t start, stop;
    hipEventCreate (&start);
	hipEventCreate (&stop);

    int numIterations = 100;
	for (int i = -1; i < numIterations; i++)
	{
		if (i == 0)
		{
			hipDeviceSynchronize();
			hipEventRecord(start, 0);
		}
	gpuRand<<<32, 128>>>(d_rand_out, N_PER_RNG);
    }

    hipEventRecord (stop, 0);
	hipEventSynchronize (stop);
	hipEventElapsedTime (&hTimer, start, stop);
    float gpuTime = 1.0e-3 * hTimer/(double)numIterations;

    fprintf(log_file, "MersenneTwister (GPU), Time = %f s, TP = %f GNumbers/s, Size = %u\n", gpuTime, 1.0e-9 * RAND_N / gpuTime, RAND_N);
    hipMemcpy(h_randGPU_out, d_rand_out, RAND_N * sizeof(float), hipMemcpyDeviceToHost);

    float hTimer_cpu;
    hipEventRecord(start, 0);
    RandomRef(h_randCPU_out, N_PER_RNG, SEED);
    hipEventRecord (stop, 0);
    hipEventElapsedTime (&hTimer_cpu, start, stop);
    float cpuTime = 1.0e-3 * hTimer_cpu/(double)numIterations;

    fprintf(log_file, "MersenneTwister (CPU), Time = %f s, TP = %f GNumbers/s, Size = %u\n", cpuTime, 1.0e-9 * RAND_N / cpuTime, RAND_N);

    hipFree(d_rand_out);
    return 0;
}
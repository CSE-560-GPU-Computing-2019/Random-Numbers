
#include <hip/hip_runtime.h>
#include <stdio.h>

typedef unsigned long long llu;

int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

#define MT_RNG_COUNT 4096

const int    PATH_N = 24000000;
const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

__global__ void RandomCT(llu *device_array, int npr) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    llu seed = 10 * tid + 5;

    llu b;
    for (int i = 0; i < npr; ++i) {
        b = (((seed << 5) ^ seed) >> 39);
        seed = (((seed & 18446744073709551614ULL) << 24) ^ b);

        device_array[tid + i * MT_RNG_COUNT] = seed;
    }

}

int main() {
    llu *device_array;
    
    llu *host_array;
    llu *host_copy;

    host_array = (llu *)malloc(RAND_N * sizeof(llu));
    host_copy = (llu *)malloc(RAND_N * sizeof(llu));
    hipMalloc((void**)&device_array, RAND_N * sizeof(llu));

    // int iters = 100;
    // for (int i = 0; )
    RandomCT<<<32, 128>>>(device_array, N_PER_RNG);

    hipMemcpy(host_copy, device_array, RAND_N * sizeof(llu), hipMemcpyDeviceToHost);
    
    // for (int i = 0; i < 10; ++i) {
    //     printf("%llu\n", host_copy[i]);
    // }

    return 0;
}
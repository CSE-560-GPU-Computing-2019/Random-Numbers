
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <sstream>
#include <iostream>
using namespace std;

typedef unsigned long long llu;

int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

#define MT_RNG_COUNT 4096

// const int    PATH_N = 1000000;
// const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
// const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

__global__ void RandomCT(llu *device_array, int npr) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    llu seed = 23 * tid + 200;
    llu b;

    for (int i = 0; i < 4; ++i) {
        b = (((seed << 5) ^ seed) >> 39);
        seed = (((seed & 18446744073709551614ULL) << 24) ^ b);
    }

    for (int i = 0; i < npr; ++i) {
        b = (((seed << 5) ^ seed) >> 39);
        seed = (((seed & 18446744073709551614ULL) << 24) ^ b);

        device_array[tid + i * MT_RNG_COUNT] = seed;
    }

}

int main() {
    // freopen("CombinedParallel.txt", "w", stdout);
    for (int i = 1000; i <= 1000; i *= 10) {
        ostringstream os;
        os << "RANDOMNUMBERS_" << i << "_CT_GPU.txt";
        string x = os.str();
        freopen(x.c_str(), "w", stdout);
        // for (int i = 10000; i <= 1000000000; i *= 10) {
        //     int    PATH_N = i;
        //     int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
        //     int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

        //     printf("%d ", RAND_N);
        // }
        // printf("\n");
        // float ktime[6], tTime[6];
        // for (int i = 10000, j = 0; i <= 1000000000, j < 6; i *= 10, j++) {
            int    PATH_N = i;
            int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
            int    RAND_N = MT_RNG_COUNT * N_PER_RNG;


            llu *device_array;
            // cudaEvent_t start, stop, memstart, memstop;
            // cudaEventCreate(&memstart);
            // cudaEventCreate(&memstop);
            // cudaEventCreate(&start);
            // cudaEventCreate(&stop);
            
            llu *host_array;
            llu *host_copy;

            host_array = (llu *)malloc(RAND_N * sizeof(llu));
            host_copy = (llu *)malloc(RAND_N * sizeof(llu));
            // cudaEventRecord(memstart);
            hipMalloc((void**)&device_array, RAND_N * sizeof(llu));

            // int iters = 100;
            // for (int i = 0; )
            // cudaEventRecord(start);
            RandomCT<<<32, 128>>>(device_array, N_PER_RNG);
            // cudaEventRecord(stop);

            hipMemcpy(host_copy, device_array, RAND_N * sizeof(llu), hipMemcpyDeviceToHost);
            // cudaEventRecord(memstop);

            // cudaEventSynchronize(stop);
            // cudaEventSynchronize(memstop);
            

            // float kernelTime, totalTime;
            // cudaEventElapsedTime(&kernelTime, start, stop);
            // cudaEventElapsedTime(&totalTime, memstart, memstop);
            // kernelTime /= 1000.0f;
            // totalTime /= 1000.0f;
            

            // cout << "Time taken for " << RAND_N << " random numbers: \n";
            // cout << "Kernel Execution time: " << kernelTime << "s\n";
            // cout << "Overall Time: " << totalTime << "s\n";
            // ktime[j] = kernelTime;
            // tTime[j] = totalTime;


            
            // for (int i = 0; i < 10; ++i) {
            //     printf("%llu\n", host_copy[i]);
            // }


            for (int j = 0; j < RAND_N; ++j) {
                printf("%llu\n", host_copy[j]);
            }

            free(host_array); free(host_copy);
            hipFree(device_array);
        // }
        // for (int i = 0; i < 6; ++i) 
        //     printf("%f ", ktime[i]);
        // printf("\n");
        // for (int i = 0; i < 6; ++i) 
        //     printf("%f ", tTime[i]);
        // printf("\n");
        fclose(stdout);
    }

    return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <iostream>

using namespace std;

#define MAXCHAR 1000
#define PI 3.141592653589793f

typedef unsigned long long llu;


__device__ double2 Box(double a, double b){
    double r = sqrt(-2*log(a));
    double theta = 2*PI*b;
    double2 xx = make_double2(r*sin(theta), r*cos(theta));
    return xx;
}


__global__ void RandomBM(double *uniform_normal_device, double *gaussian_device){
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    double2 uni = Box(uniform_normal_device[tid*2], uniform_normal_device[tid*2 + 1]);
    gaussian_device[tid*2] = uni.x;
    gaussian_device[tid*2 + 1] = uni.y;
}


int main(){
    // freopen("BoxMullerParallel.txt", "w", stdout);

    float ktime[6], tTime[6];
    int q = 0;
    for(int i = 512; i<=512; i*=2){
        printf("%d ", i);
        printf("\n");
        hipEvent_t start, stop, memstart, memstop;
        hipEventCreate(&memstart);
        hipEventCreate(&memstop);
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int X = 256;
        int Y = i;
        llu RAND_N = X*Y;
        FILE *fp = fopen("RANDOMNUMBERS_HT_GPU.txt", "r");
        llu num;
        llu *uniform;
        double *uniform_normal_device;
        double *uniform_normal_host;
        double *gaussian_device;
        double *gaussian_host;

        uniform = (llu *)malloc(RAND_N * sizeof(llu));
        uniform_normal_host = (double *)malloc(RAND_N * sizeof(double));
        gaussian_host = (double *)malloc(RAND_N * sizeof(double));

        hipEventRecord(memstart);

        hipMalloc((void**)&uniform_normal_device, RAND_N * sizeof(double));
        hipMalloc((void**)&gaussian_device, RAND_N * sizeof(double));


        llu counter = 0;
        llu max = 0;

        while (fscanf(fp,"%llu",&num) != EOF && counter < RAND_N){
            if(num > max) max = num;
            uniform[counter] = num;
            // printf("%llu\n", num);
            counter++;
        }
        // printf("Counter: %llu\n",counter);
        // printf("MAX: %llu\n",max);

        for(int i = 0; i<counter; i++) uniform_normal_host[i] = uniform[i]/double(max);
        // printf("--------------\n\n");
        // for(int i = 0; i<counter; i++) printf("%.17g\n", uniform_normal_host[i]); 
        // printf("--------------\n\n");

        hipMemcpy(uniform_normal_device, uniform_normal_host, RAND_N * sizeof(double), hipMemcpyHostToDevice);

        free(uniform);

        hipEventRecord(start);
        RandomBM<<<X, Y/2>>>(uniform_normal_device, gaussian_device);
        hipEventRecord(stop);    

        hipMemcpy(gaussian_host, gaussian_device, RAND_N * sizeof(double), hipMemcpyDeviceToHost);

        hipEventRecord(memstop);

        hipEventSynchronize(stop);
        hipEventSynchronize(memstop);


        // WRITE TO A FILE
        FILE *F;
        F = freopen("NORMAL_RANDOMNUMBERS_BOX_GPU.txt", "w", stdout);
        for(int i = 0; i<counter; i++) printf("%.17g\n", gaussian_host[i]); 
        fclose(F);

        float kernelTime, totalTime;
        hipEventElapsedTime(&kernelTime, start, stop);
        hipEventElapsedTime(&totalTime, memstart, memstop);
        kernelTime /= 1000.0f;
        totalTime /= 1000.0f;

        cout << "Time taken for " << RAND_N << " random numbers: \n";
        cout << "Kernel Execution time: " << kernelTime << "s\n";
        cout << "Overall Time: " << totalTime << "s\n";
        ktime[q] = kernelTime;
        tTime[q] = totalTime;

        hipFree(gaussian_device);
        hipFree(uniform_normal_device);

        free(gaussian_host);
        free(uniform_normal_host);

        fclose(fp);
        q++;
    }
    for (int i = 0; i < 6; ++i) 
        printf("%f ", ktime[i]);
    printf("\n");
    for (int i = 0; i < 6; ++i) 
        printf("%f ", tTime[i]);
    printf("\n");

}
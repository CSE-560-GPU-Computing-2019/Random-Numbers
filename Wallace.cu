#include "hip/hip_runtime.h"
#include <stdio.h>

POOL_PASSES = 10
A = 1664525 
B = 1013904223ULL

void Transform(){
    // K, and M are binary powers.
    const unsigned K = 128  // Size of pool
    const unsigned M = K/D;  // Number of threads, and LCG modulus
    float block_0, block_1 , block_2 , block_3; 
    for(int pass = 0; pass < POOL_PASSES; pass++){
        // Read the pool in using a pseudorandom permutation.
        unsigned s = tid;
         // M is a binary power, don't need %.
        // s is being recomputed as an LCG.
        s = (s*A+B) & (M-1); block_0=pool[(s<<3)+0];
        s = (s*A+B) & (M-1); block_1=pool[(s<<3)+1];
        s = (s*A+B) & (M-1); block_2=pool[(s<<3)+2];
        s = (s*A+B) & (M-1); block_3=pool[(s<<3)+3];
        // All pool values must be read before any are written.
        __syncthreads();
        // Perform in-place 4x4 orthogonal transform on block.
        TransformBlock(block);
        // Output the blocks in linear order.
        s=tid;
        pool[s]=block_0; s+=NT;
        pool[s]=block_1; s+=NT;
        pool[s]=block_2; s+=NT;
        pool[s]=block_3; s+=NT;
    }
}

__device__ void TransformBlock(float *b){
  float t=(b[0]+b[1]+b[2]+b[3])/2;
  b[0]=b[0]-t;
  b[1]=b[1]-t;
  b[2]=t-b[2];
  b[3]=t-b[3];
}


__device__ void generateRandomNumbers_wallace(
    unsigned seed,  // Initialization seed
    float *chi2Corrections,  // Set of correction values
    float *globalPool,  // Input random number pool
    float *output  // Output random numbers
    ){
    unsigned tid=threadIdx.x;
    // Load global pool into shared memory.
    unsigned offset = POOL_SIZE * blockIdx.x;
    for( int i = 0; i < 4; i++) pool[tid+THREADS*i] = globalPool[offset+TOTAL_THREADS*i+tid];
    __syncthreads();
      const unsigned lcg_a=241;
      const unsigned lcg_c=59;
      const unsigned lcg_m=256;
      const unsigned mod_mask = lcg_m-1;
      seed=(seed+tid)&mod_mask ;
      // Loop generating outputs repeatedly
    for( int loop = 0; loop < OUTPUTS_PER_RUN; loop++ ){
        Transform();
        unsigned intermediate_address;
        i_a = loop * 8 * TOTAL_THREADS) + 8 * THREADS * blockIdx.x + threadIdx.x;
        float chi2CorrAndScale=chi2Corrections[
          blockIdx.x * OUTPUTS_PER_RUN + loop];
        for( i = 0; i < 4; i++ )
          output[i_a + i*THREADS]=chi2CorrAndScale*pool[tid+THREADS*i];
    }
  }
  